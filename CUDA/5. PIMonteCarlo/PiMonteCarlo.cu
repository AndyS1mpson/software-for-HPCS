#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <ctime>

using namespace std;
const int Radius = 1000;
const int N = 100000;

__global__ void PiCuda(double* points, int* count)
{
	//__shared__ int cCount[N];
	//for (int i = 0; i < N; i++)
		//cCount[i] = 0;
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	if (points[tx * 2] * points[tx * 2] + points[tx * 2 + 1] * points[tx * 2 + 1] < Radius * Radius)
		atomicAdd(count, 1);
	//cCount[tx]++;
//__syncthreads();
//count[0] += cCount[tx];
}

int Pi(double* points)
{
	int count = 0;
	for (int i = 0; i < N; i++)
	{
		if (points[i * 2] * points[i * 2] + points[i * 2 + 1] * points[i * 2 + 1] < Radius * Radius)
			count++;
	}

	return count;
}



void CreatePoints(double* Points)
{
	srand(time(0));
	for (int i = 0; i < N; i++)
	{
		Points[i * 2] = rand() % Radius;
		Points[i * 2 + 1] = rand() % Radius;
	}
}

int main()
{
	//Переменные для измерения времени выполнения CUDA.
	hipEvent_t start, end;
	float Time = 0;

	double points[N * 2];
	CreatePoints(points);
	int countPointsInCircle = 0;

	double* cudaPoints;
	int* cudaCountPointsInCircle;
	hipMalloc((void**)&cudaPoints, sizeof(double) * N * 2);
	hipMalloc((void**)&cudaCountPointsInCircle, sizeof(int));

	hipMemcpy(cudaPoints, &points, sizeof(double) * N * 2, hipMemcpyHostToDevice);
	hipMemcpy(cudaCountPointsInCircle, &countPointsInCircle, sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&end);
	
	// Запуск таймера
	hipEventRecord(start);
	float t_start = clock();
	
	countPointsInCircle = Pi(points);
	float t_end = clock();
	cout << "Время работы последовательной версии:" << t_end - t_start << endl;
	// Остановка таймера
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&Time, start, end);
	
	cout << "PI = " << 4 * double(countPointsInCircle) / N << endl;

	dim3 blocks = 100;
	dim3 threads = 1000;
	// Запуск таймера
	hipEventRecord(start);
	PiCuda << < blocks, threads >> > (cudaPoints, cudaCountPointsInCircle);
	// Остановка таймера
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&Time, start, end);
	cout << "Время работы на GPU:" << Time << endl;
	hipDeviceSynchronize();
	hipMemcpy(&countPointsInCircle, cudaCountPointsInCircle, sizeof(int), hipMemcpyDeviceToHost);
	cout << "PI = " << 4 * float(countPointsInCircle) / float(N) << endl;

	hipFree(cudaPoints);

	return 0;
}