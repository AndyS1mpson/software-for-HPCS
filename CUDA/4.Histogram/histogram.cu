#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <fstream>
#include <cstring>
#include <ctime>

using namespace std;

const int countLetter = 26;
const int lenLine = 1024;


__global__ void SumLetterCuda(int* gist, char* line, char* letter)
{
	int i = blockIdx.x;  //countLetter = 26
	int j = threadIdx.x; // lenLine = 1024
	if (letter[i] == line[j])
		atomicAdd(&gist[i], 1);
}

void SumLetter(int* gist, char* line, char* letter)
{
	for (int i = 0; i < lenLine; i++) // 1024
	{
		for (int j = 0; j < countLetter; j++) //26
		{
			if (line[i] == letter[j])
				gist[j]++;
		}
	}
}

void PrintMas(int* mas, int N)
{
	for (int i = 0; i < N; i++) {
		cout << mas[i] << " ";
	}
	cout << endl;
}

void PrintMasChar(char* mas, int N)
{
	for (int i = 0; i < N; i++) {
		cout << mas[i] << " ";
	}
	cout << endl;
}


int main()
{
	float Time = 0;

	int* gist = new int[countLetter]; //26
	for (int i = 0; i < countLetter; i++)
		gist[i] = 0;
	char letter[countLetter] = { 'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z' };
	char buff[lenLine]; //1024

	ifstream fin("text.txt");
	fin.getline(buff, lenLine);
	fin.close();

	Time = clock();
	SumLetter(gist, buff, letter);
	cout << "Время выполнения на CPU " << Time << endl;

	cout << "Результат работы:" << endl;
	PrintMasChar(letter, countLetter);
	PrintMas(gist, countLetter);

	int* cudaGist;
	char* cudaLetter;
	char* cudaBuff;
	
	// Переменные для измерения времени выполнения CUDA.
	Time = 0;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	// Выделение памяти
	hipMalloc((void**)&cudaGist, sizeof(int) * countLetter);
	hipMalloc((void**)&cudaLetter, sizeof(char) * countLetter);
	hipMalloc((void**)&cudaBuff, sizeof(char) * lenLine);

	hipMemcpy(cudaGist, gist, sizeof(int) * countLetter, hipMemcpyHostToDevice);
	hipMemcpy(cudaLetter, letter, sizeof(char) * countLetter, hipMemcpyHostToDevice);
	hipMemcpy(cudaBuff, buff, sizeof(char) * lenLine, hipMemcpyHostToDevice);

	dim3 blocks = countLetter;
	dim3 threads = lenLine;

	// Запуск таймера
	hipEventRecord(start);
	
	SumLetterCuda << < blocks, threads >> > (cudaGist, cudaBuff, cudaLetter);
	hipMemcpy(gist, cudaGist, sizeof(int) * countLetter, hipMemcpyDeviceToHost);

	// Остановка таймера
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&Time, start, end);
	cout << "Время работы на GPU " << Time << endl;
	cout << "Результат работы:" << endl;
	PrintMasChar(letter, countLetter);
	PrintMas(gist, countLetter);

	hipFree(cudaGist);
	hipFree(cudaBuff);
	hipFree(cudaLetter);

	return 0;
}