
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <stdlib.h>

#define N   10

__global__
void add( int *a, int *b, int *c ) {
    int tid = 0;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += 1;
    }
}

int main( void ) {
    size_t size = N* sizeof(int);

    int* h_a = (int*)malloc(size);
    int* h_b = (int*)malloc(size);
    int* h_c = (int*)malloc(size);
    for (int i=0; i<N; i++) {
        h_a[i] = -i;
        h_b[i] = i * i;
    }

    int* d_a;
    int* d_b;
    int* d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 1;
    int blockPerGrid = 1;

    add <<<blockPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", h_a[i], h_b[i], h_c[i] );
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
