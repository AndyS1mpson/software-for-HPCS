
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <stdlib.h>

#define N   10

__global__
void add( int *a, int *b, int *c ) {
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

int main( void ) {
    size_t size = N* sizeof(int);

    int* h_a = (int*)malloc(size);
    int* h_b = (int*)malloc(size);
    int* h_c = (int*)malloc(size);

    for (int i=0; i<N; i++) {
        h_a[i] = -i;
        h_b[i] = i * i;
    }

    int* d_a;
    int* d_b;
    int* d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int blockPerGrid = 1;
    int threadsPerBlock = N;

    add <<<blockPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", h_a[i], h_b[i], h_c[i] );
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

