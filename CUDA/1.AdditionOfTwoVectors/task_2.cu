
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <stdlib.h>
#define Num_elements 129

__global__
void add( int *a, int *b, int *c) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < Num_elements)
        c[tid] = a[tid] + b[tid];
}

int main( void ) {

    size_t size = Num_elements* sizeof(int);

    int* h_a = (int*)malloc(size);
    int* h_b = (int*)malloc(size);
    int* h_c = (int*)malloc(size);
    for (int i=0; i<Num_elements; i++) {
        h_a[i] = -i;
        h_b[i] = i * i;
    }

    int* d_a;
    int* d_b;
    int* d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int Num_threads = 8;

    add <<< (Num_elements + (Num_threads - 1))/Num_threads , Num_threads >>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i=0; i<Num_elements; i++) {
        printf( "%d + %d = %d\n", h_a[i], h_b[i], h_c[i] );
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
